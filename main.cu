#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <typeinfo>
#include <time.h>
#include <opencv2/opencv.hpp>

// command line args for opencv
// -lopencv_core -lopencv_imgcodecs

// main command: nvcc main.cu -lopencv_core -lopencv_imgcodecs && ./a.out img1.jpg

using namespace cv;
using namespace std;

struct RLE
{
  int value;
  int count;
};

int pairingCPU(int x, int y)
{
  return (x>=y) ? (x*x + x + y) : (y*y + x);
}

RLE unpairingCPU(int z)
{
  struct RLE res;
  int b = (int)sqrt(z);
  int a = z - b*b;
  if(a<b)
  {
    res.value=a;
    res.count=b;
  }
  else
  {
    res.value=b;
    res.count=a-b;
  }
  return res;
}

__global__ void pairingGPU(const RLE *rle, int *res, int numElements) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < numElements) {
    int x = rle[i].value;
    int y = rle[i].count;
    res[i] = (x>=y) ? (x*x + x + y) : (y*y + x);
  }
}

__global__ void unpairingGPU(const int *compressed, RLE *decompressed, int numElements) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < numElements) {
    int z = *(compressed+i);
    int b = (int)sqrt((float)z);
    int a = z - b*b;

    RLE rle;
    if(a<b)
    {
      rle.value = a;
      rle.count = b;
    }
    else
    {
      rle.value = b;
      rle.count = a-b;
    }
    *(decompressed+i) = rle;
  }
}

void printVectorInt(vector<int> v, int n)
{
  for(int i=0; i<n; i++)
    cout << v[i] << " ";
  cout << endl;
}

void printVectorRLE(vector<RLE> v, int n)
{
  for(int i=0; i<n; i++)
    cout << "[" << v[i].value << "," << v[i].count << "] ";
  cout << endl;
}

void printArrayInt(int *v, int n)
{
  for(int i=0; i<n; i++)
    cout << *(v+i) << " ";
  cout << endl;
}

void printArrayRLE(RLE *v, int n)
{
  for(int i=0; i<n; i++)
  {
    RLE x = *(v+i);
    cout << "[" << x.value << "," << x.count << "] ";
  }
  cout << endl;
}

int main(int argc, char *argv[])
{
  clock_t begin, end;
  double time_spent1, time_spent2;

  // Load the image using OpenCV
  Mat image = imread(argv[1], cv::IMREAD_GRAYSCALE);
  if (image.empty()) {
    cout << "Failed to load image!" << endl;
    return 1;
  }

  vector<RLE> compressed;
  int count = 0;
  int prevPixel = static_cast<int>(image.at<unsigned char>(0,0));
  for (int row = 0; row < image.rows; row++) {
    for (int col = 0; col < image.cols; col++) {
      int currPixel = static_cast<int>(image.at<unsigned char>(row,col));
      if(currPixel == prevPixel)
        count++;
      else {
        struct RLE rle;
        rle.value = currPixel;
        rle.count = count;

        compressed.push_back(rle);

        prevPixel = currPixel;
        count = 1;
      }
    }
  }


  // CPU
  cout << endl << "```````CPU```````" << endl;
  cout << "after compression:            ";
  printVectorRLE(compressed, 5);

  // further compression by using pairing function
  begin = clock();
  vector<int> enhCompressed;
  for(auto item : compressed)
    enhCompressed.push_back(pairingCPU(item.value, item.count));
  end = clock();
  time_spent1 = (double)(end - begin) / CLOCKS_PER_SEC;
  cout << "after enhanced compression:   ";
  printVectorInt(enhCompressed, 5);
  
  // do decompression
  begin = clock();
  vector<RLE> enhDecompressed;
  for(auto item : enhCompressed)
    enhDecompressed.push_back(unpairingCPU(item));
  end = clock();
  time_spent2 = (double)(end - begin) / CLOCKS_PER_SEC;
  cout << "after enhanced decompression: ";
  printVectorRLE(enhDecompressed, 5);
  
  // compare the decompressed and original versions, should be 100% same
  // i.e. compare compressed and enhDecompressed vectors
  int cnt1=0;
  for(int i=0; i<enhDecompressed.size(); i++)
    if(enhDecompressed[i].value==compressed[i].value && enhDecompressed[i].count==compressed[i].count)
      cnt1++;
  cout << endl << "Percentage of same elements: " << ((cnt1)/(float)(compressed.size()))*100.0 << endl;

  cout << fixed;
  cout.precision(8);
  cout << "Time for compression:   " << time_spent1 << " seconds" << endl;
  cout << "Time for decompression: " << time_spent2 << " seconds" << endl;
  cout.precision(0);


  // GPU
  cout << endl << "```````GPU```````" << endl;
  cout << "after compression:            ";
  printVectorRLE(compressed, 5);

  // GPU datastructures initialization and stuff
  int size1 = compressed.size();
  int threadsPerBlock = 8;
  int blocksPerGrid = size1;
  // 3 vectors in host   (CPU) - compressedC, enhCompressedC, enhDecompressedC
  RLE *compressedC      = (RLE *)malloc(size1*sizeof(RLE));
  int *enhCompressedC   = (int *)malloc(size1*sizeof(int));
  RLE *enhDecompressedC = (RLE *)malloc(size1*sizeof(RLE));
  // initialize them
  for(int i=0; i<size1; i++)
  {
    compressedC[i].value = compressed[i].value;
    compressedC[i].count = compressed[i].count;
    
    enhCompressedC[i] = 0;
    enhDecompressedC[i].value = 0;
    enhDecompressedC[i].count = 0;
  }
  // 3 vectors in device (GPU) - compressedG, enhCompressedG, enhDecompressedG
  RLE *compressedG      = NULL;
  int *enhCompressedG   = NULL;
  RLE *enhDecompressedG = NULL;
  hipMalloc((void **)&compressedG     , size1*sizeof(RLE));
  hipMalloc((void **)&enhCompressedG  , size1*sizeof(int));
  hipMalloc((void **)&enhDecompressedG, size1*sizeof(RLE));
  // copy host vector to device vector
  hipMemcpy(compressedG, compressedC, size1*sizeof(RLE), hipMemcpyHostToDevice);

  // further compression by using pairing function
  begin = clock();
  pairingGPU<<<blocksPerGrid, threadsPerBlock>>>(compressedG, enhCompressedG, size1);
  end = clock();
  time_spent1 = (double)(end - begin) / CLOCKS_PER_SEC;
  // copy back to host and print it
  hipMemcpy(enhCompressedC, enhCompressedG, size1*sizeof(int), hipMemcpyDeviceToHost);
  cout << "after enhanced compression:   ";
  printArrayInt(enhCompressedC, 5);

  // do decomposition
  begin = clock();
  unpairingGPU<<<blocksPerGrid, threadsPerBlock>>>(enhCompressedG, enhDecompressedG, size1);
  end = clock();
  time_spent2 = (double)(end - begin) / CLOCKS_PER_SEC;
  // copy back to host and print it
  hipMemcpy(enhDecompressedC, enhDecompressedG, size1*sizeof(RLE), hipMemcpyDeviceToHost);
  cout << "after enhanced decompression: ";
  printArrayRLE(enhDecompressedC, 5);

  // compare the decompressed and original versions, should be 100% same
  // i.e. compare compressedC and enhDecompressedC vectors
  cnt1=0;
  for(int i=0; i<size1; i++)
  {
    RLE rle1, rle2;
    rle1 = *(enhDecompressedC+i);
    rle2 = *(compressedC+i);
    if(rle1.value==rle2.value && rle1.count==rle2.count)
      cnt1++;
  }
  cout << endl << "Percentage of same elements: " << ((cnt1)/(float)(size1))*100.0 << endl;

  cout << fixed;
  cout.precision(8);
  cout << "Time for compression:   " << time_spent1 << " seconds" << endl;
  cout << "Time for decompression: " << time_spent2 << " seconds" << endl;
  cout.precision(4);


  // Some comparisions
  cout << endl << "```````STATISTICS```````" << endl;
  int originalSize = image.rows * image.cols * 1; // 8-bit image (8 bits = 1 byte)
  int compressedSize = (1+sizeof(int)) * compressed.size(); // 1 btye for color value + 1 int for count
  int enhCompressedSize = sizeof(int) * compressed.size(); // 1 int
  cout << "original size of image:   " << originalSize      << " bytes" << endl;
  cout << "compressed size:          " << compressedSize    << " bytes" << endl;
  cout << "enhanced compressed size: " << enhCompressedSize << " bytes" << endl;
  cout << endl;
  cout << "compression ratio (org|compr):       " << ((originalSize)/(float)(compressedSize)) << endl;
  cout << "compression ratio (compr|enh_compr): " << ((compressedSize)/(float)(enhCompressedSize)) << endl;

  cnt1 = 0;
  for(auto item : compressed)
    if (item.count>1)
      cnt1++;
  cout << "Found " << cnt1 << " repetitive sequences, image size = " << image.rows << " X " << image.cols
       << " (" << (image.rows*image.cols) << " pixels)" << endl;

  return 0;
}

